#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "tensorview.h"

struct LaunchParams
{
    opg::TensorView<float, 1> output;
};

__constant__ LaunchParams params;

extern "C" __global__ void __miss__nop()
{
}

extern "C" __global__ void __raygen__main()
{
    //printf("\nBefore write %p\n", params.output.data);

    params.output[0].value() = 1.0f;

    printf("After write\n");
}
